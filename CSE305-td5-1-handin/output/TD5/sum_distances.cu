
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <cmath>

__device__
double DistKer(double* p, double* q, size_t dim) {
    double result = 0;
    for (size_t i = 0; i < dim; ++i) {
        result += (p[i] - q[i]) * (p[i] - q[i]);
    }
    return std::sqrt(result);
}

double Dist(double* p, double* q, size_t dim) {
    double result = 0;
    for (size_t i = 0; i < dim; ++i) {
        result += (p[i] - q[i]) * (p[i] - q[i]);
    }
    return std::sqrt(result);
}

//------------------------------------------------

double SumDistances(double* arr, size_t dim, size_t N) {
    double result = 0.;
    for (size_t i = 0; i < N; ++i) {
        double* p = arr + i * dim;
        for (size_t j = i + 1; j < N; ++j) {
            result += Dist(p, arr + j * dim, dim);
        }
    }
    return result;
}

//-------------------------------------------------

__global__
void SumDistancesGPUAux(double* arr, size_t dim , size_t N, double* results_gpu) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) {
        return;
    }
    double result = 0.;
    double* p = arr + index * dim;
    for (size_t i = index + 1; i < N; ++i) {
        result += DistKer(p, arr + i * dim, dim);
    }
    results_gpu[index] = result;
}

double SumDistancesGPU(double* arr, size_t dim, size_t N) {
    const size_t THREADS_PER_BLOCK = 256;
 
    // moving the input to the device
    double* arr_device;
    hipMalloc(&arr_device, N * dim * sizeof(double));
    hipMemcpy(arr_device, arr, N * dim * sizeof(double), hipMemcpyHostToDevice);

    // allocating memory for the output
    double* results_gpu;
    hipMalloc(&results_gpu, N * sizeof(double));

    // running calculation on GPU
    size_t blocks_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    SumDistancesGPUAux<<<blocks_num, THREADS_PER_BLOCK>>>(arr_device, dim, N, results_gpu);
    hipDeviceSynchronize();

    // Computing the final result
    double* results_cpu = (double*) malloc(N * sizeof(double));
    hipMemcpy(results_cpu, results_gpu, N * sizeof(double), hipMemcpyDeviceToHost);

    double final_result = 0;
    for (size_t i = 0; i < N; ++i) {
        final_result += results_cpu[i];
    }

    // Freeing the memory
    hipFree(arr_device);
    hipFree(results_gpu);
    free(results_cpu);
    return final_result; 
}

//---------------------------------------------------

__global__
void SumDistancesGPUAux2(double* arr, size_t dim , size_t N, double* results_gpu) {
    extern __shared__ double bf[];
    size_t index = blockIdx.x * blockDim.x * threadIdx.x;

    if (index >= N){
        return; 
    }

    for (size_t i = 0; i < dim; ++i){ 
        bf[threadIdx.x + i] = arr[index * dim + i];
    }
    __syncthreads();
    double* p = &bf[threadIdx.x];
    double result = 0.; 

    for (size_t i = threadIdx.x; i < blockDim.x; ++i){ 
        double* q = &bf[threadIdx.x + i * dim]; 
        result += DistKer(p, q, dim);
    }


    for (size_t b = blockIdx.x; b < gridDim.x; ++b){ 
        if (b != blockIdx.x){
            for (size_t thread_num = 0; thread_num < blockDim.x * dim; ++thread_num){ 
                for (size_t d = 0; d < dim; ++d){
                    bf[blockDim.x + (thread_num * dim) + d] = arr[(b * blockDim.x * dim) + (thread_num * dim) + d];
                }
            }
            __syncthreads();
            for (size_t thread_num = 0; thread_num < blockDim.x * dim; ++thread_num){ 
                double* q = &bf[blockDim.x + (thread_num * dim)];
                result += DistKer(p, q, dim);
            }
        }
    }
}

double SumDistancesGPU2(double* arr, size_t dim, size_t N) {
    const size_t TOTAL_THREADS = N;
    const size_t THREADS_PER_BLOCK = 64;
    size_t NUM_BLOCKS = (TOTAL_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // moving the input to the device
    double* arr_device;
    hipMalloc(&arr_device, N * dim * sizeof(double));
    hipMemcpy(arr_device, arr, N * dim * sizeof(double), hipMemcpyHostToDevice);

    // allocating memory for the output
    double* results_gpu;
    hipMalloc(&results_gpu, N * sizeof(double));

    // running calculation on GPU
    SumDistancesGPUAux2<<<NUM_BLOCKS, THREADS_PER_BLOCK, THREADS_PER_BLOCK * dim * sizeof(double)>>>(arr_device, dim, N, results_gpu);
    hipDeviceSynchronize(); 

    // Computing the final result
    double* results_cpu = (double*) malloc(N * sizeof(double));
    hipMemcpy(results_cpu, results_gpu, N * sizeof(double), hipMemcpyDeviceToHost);

    double final_result = 0;
    for (size_t i = 0; i < N; ++i) {
        final_result += results_cpu[i];
    }

    // Freeing the memory
    hipFree(arr_device);
    hipFree(results_gpu);
    free(results_cpu);
    return final_result; 
 
    return 0;
}

//---------------------------------------------------

int main(int argc, char* argv[]) {
    // setting the random seed to get the same result each time
    srand(42);

    // taking as input, which algo to run
    int alg_ind = std::stoi(argv[1]);

    // Generating data
    size_t N = 6400;
    size_t dim = 3;
    double* arr = (double*) malloc(N * dim * sizeof(double));
    for (size_t i = 0; i < dim * N; ++i) {
          arr[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
    }
 
    // Computing on CPU
    double result = 0.;
    auto start = std::chrono::steady_clock::now();
    switch (alg_ind) {
        case 0: 
            result = SumDistances(arr, dim, N);
            break;
        case 1:
            result = SumDistancesGPU(arr, dim, N);
            break;
        case 2:
            result = SumDistancesGPU2(arr, dim, N);
            break;
    }
    auto finish = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
    std::cout << "Elapsed time: " << elapsed << std::endl;
    std::cout << "Total result: " << result << std::endl;
    
    free(arr);
    return 0;
}
