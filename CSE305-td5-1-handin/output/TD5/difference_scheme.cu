
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>

const double C = 0.5;

//------------------------------------------------

void SolvePDE(double* boundary_values, size_t N, double dx, double dt, size_t timesteps, double* result) {
    double* curr = (double*) malloc(N * sizeof(double));
    double* next = (double*) malloc(N * sizeof(double));
    memcpy(curr, boundary_values, N * sizeof(double));
    for (size_t i = 0; i < timesteps; ++i) {
        for (size_t j = 0; j < N; ++j) {
            if (j < N - 1) {
                next[j] = curr[j] + C * (dt / dx) * (curr[j + 1] - curr[j]);
            } else {
                next[j] = curr[j] + C * (dt / dx) * (curr[0] - curr[j]);
            }
        }
        std::swap(curr, next);
    }
    memcpy(result, curr, N * sizeof(double));
    free(curr);
    free(next);
}

//-------------------------------------------------

//CUDA kernel to perform the PDE computation on the GPU
__global__
void PDEAux(double* curr, double* next, size_t N, double dx, double dt){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;  //index of thread
    if (index >= N) {
        return;
    }
    //if not last element
    if (index < N - 1) {
        next[index] = curr[index] + C * (dt / dx) * (curr[index+1] - curr[index]);
    }
    else {
        next[index] = curr[index] + C * (dt / dx) * (curr[0] - curr[index]); 
    }
}

// __global__
// void SolvePDEGPUAux(double* curr, double* next, size_t N, double dx, double dt){
//     size_t index = blockIdx.x * blockDim.x + threadIdx.x;  
//     if (index >= N) {
//         return;
//     }
//     if (index < N - 1) {
//     
//     }
// }

/**
 * @brief Solves a PDE u_t = C * u_x using a simple difference scheme
 * @param boundary_values - the pointer to the beginning of an array of values at t = 0
 * @param N - the length of the array
 * @param dx - step size for x coordinate
 * @param dt - step size for t coordinate
 * @param timesteps - number of steps in time to preform
 * @param result - pointer to yhe array for the value at the last time step
 */

void SolvePDEGPU(double* boundary_values, size_t N, double dx, double dt, size_t timesteps, double* result) {
    const size_t THREADS_PER_BLOCK = 64; 
    const size_t TOTAL_THREADS = N; 

    double *currd; //to the current array of values
    double *nextd; //to the next array of values
    hipMalloc(&currd, N * sizeof(double));
    hipMalloc(&nextd, N * sizeof(double));
    hipMemcpy(currd, boundary_values, N * sizeof(double), hipMemcpyHostToDevice);

    //number of blocks needed to run all threads
    const size_t NUM_BLOCKS = (TOTAL_THREADS + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK; 
    //const size_t NUM_BLOCKS = (TOTAL_THREADS - 1)/THREADS_PER_BLOCK; 

    for (size_t t = 0; t < timesteps; ++t){
        //calling the kernel function 
        PDEAux<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(currd, nextd, N, dx, dt);
        hipDeviceSynchronize();
        std::swap(currd, nextd);
    }

    hipMemcpy(result, currd, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(currd);
    hipFree(nextd);
}

//---------------------------------------------------

__global__
void PDEAux2(double* curr, double* next, size_t N, double dx, double dt){
    extern __shared__ double bf[]; //buffer
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= N) {
        return;
    }
    bf[threadIdx.x] = curr[index];
    __syncthreads();

    if (index < N - 1){
        if (threadIdx.x < blockDim.x - 1) {
            next[index] = bf[threadIdx.x] + C * (dt / dx) * (bf[threadIdx.x + 1] - bf[threadIdx.x]);  //compute next element value
        }
        else {
            //in block - last thread
            next[index] = bf[threadIdx.x] + C * (dt / dx) * (curr[index+1] - bf[threadIdx.x]); 
        }
    }
    else{
        //in array - last thread
        next[index] = bf[threadIdx.x] + C * (dt / dx) * (curr[0] - bf[threadIdx.x]);
    }
}

// __global__
// void PDEAux2(double* curr, double* next, size_t N, double dx, double dt){
//     extern __shared__ double bf[]; 
//     size_t index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index >= N) {
//         return;
//     }
//     bf[threadIdx.x] = curr[index];
//     __syncthreads();

//     if (threadIdx.x < blockDim.x - 1) {
//         next[index] = bf[threadIdx.x] + C * (dt / dx) * (bf[threadIdx.x + 1] - bf[threadIdx.x]); 
//     }
//     else {
//         next[index] = bf[threadIdx.x] + C * (dt / dx) * (curr[index+1] - bf[threadIdx.x]); 
//     }
// }

/**
 * @brief Solves a PDE u_t = C * u_x using a simple difference scheme
 * @param boundary_values - the pointer to the beginning of an array of values at t = 0
 * @param N - the length of the array
 * @param dx - step size for x coordinate
 * @param dt - step size for t coordinate
 * @param timesteps - number of steps in time to preform
 * @param result - pointer to yhe array for the value at the last time step
 */

void SolvePDEGPU2(double* boundary_values, size_t N, double dx, double dt, size_t timesteps, double* result) {
    const size_t THREADS_PER_BLOCK = 64;
    const size_t TOTAL_THREADS = N;
    const size_t NUM_BLOCKS = (TOTAL_THREADS + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK; 

    double *currd; 
    double *nextd; 
    hipMalloc(&currd, N * sizeof(double));
    hipMalloc(&nextd, N * sizeof(double));
    hipMemcpy(currd, boundary_values, N * sizeof(double), hipMemcpyHostToDevice);

    for (size_t t = 0; t < timesteps; ++t){
        PDEAux2<<<NUM_BLOCKS, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(double)>>>(currd, nextd, N, dx, dt);
        hipDeviceSynchronize();
        std::swap(currd, nextd);  //swap the current and next arrays for the next time step
    }

    hipMemcpy(result, currd, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(currd);
    hipFree(nextd);
}

//---------------------------------------------------

int main(int argc, char* argv[]) {
    // setting the random seed to get the same result each time
    srand(42);

    // taking as input, which algo to run
    int alg_ind = std::stoi(argv[1]);

    // Generating data
    double length = 8 * atan(1.0); // 2 pi
    double dx = 0.0001;
    double dt = 0.0001;
    size_t N = int(length / dx);

    double* boundary = (double*) malloc(N * sizeof(double));
    double* result = (double*) malloc(N * sizeof(double));
    for (size_t i = 0; i < N; ++i) {
        boundary[i] = sin(i * dx);
    }

    size_t timesteps = 10000;
    auto start = std::chrono::steady_clock::now();
    switch (alg_ind) {
        case 0: 
            SolvePDE(boundary, N, dx, dt, timesteps, result);
            break;
        case 1:
            SolvePDEGPU(boundary, N, dx, dt, timesteps, result);
            break;
        case 2:
            SolvePDEGPU2(boundary, N, dx, dt, timesteps, result);
            break;
    }
    auto finish = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
   
    for (size_t i = 0; i <  N; ++i) {
        std::cout << result[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "Elapsed time: " << elapsed << std::endl;
 
    free(boundary);
    free(result);
    return 0;
}
